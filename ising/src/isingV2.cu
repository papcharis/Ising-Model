/*
* FILE: isingV2.cu
* THMMY, 7th semester, Parallel and Distributed Systems: 3rd assignment
* Parallel Implementation  of the Ising Model
* Authors:
*   Moustaklis Apostolos, 9127, amoustakl@ece.auth.gr
*   Papadakis Charis , 9128, papadakic@ece.auth.gr
* Compile command with :
*   make all
* Run command example:
*   ./src/isingV2
* It will calculate the evolution of the ising Model
* for a given number n  of points and k steps
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

// Defines for the block and grid calculation
#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16
//The dimensions of the lattice
#define N_X 517
#define N_Y 517

// The size of the weights
#define WSIZE 5

//Helper Defines to access easier the arrays
#define old(i,j,n) *(old+i*n+j)
#define current(i,j,n) *(current+i*n+j)
#define w(i,j) *(w+i*5+j)
#define d_w(i,j) *(d_w+i*5+j)
#define G(i,j,n) *(G+i*n+j)
#define d_current(i,j,n) *(d_current+i*n+j)
#define d_old(i,j,n) *(d_old+i*n+j)

//Functions Declaration
void swapElement(int  ** one, int  ** two);
__global__
   void kernel2D(int *d_current, int *d_old, double *d_w, int n , int * d_flag);
void ising( int *G, double *w, int k, int n);


//! Ising model evolution
/*!

  \param G      Spins on the square lattice             [n-by-n]
  \param w      Weight matrix                           [5-by-5]
  \param k      Number of iterations                    [scalar]
  \param n      Number of lattice points per dim        [scalar]

  NOTE: Both matrices G and w are stored in row-major format.
*/

void ising( int *G, double *w, int k, int n){

  //Grid and block construction
  dim3 block(BLOCK_SIZE_X,BLOCK_SIZE_Y);
  int grid_size_x  = (N_X + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X;
  int grid_size_y  = (N_Y + BLOCK_SIZE_Y - 1) / BLOCK_SIZE_Y;
  dim3 grid(grid_size_x,grid_size_y);
  //Device memory allocation
  int * old = (int*) malloc(n*n*(size_t)sizeof(int)); // old spin lattice
  int * current = (int*) malloc(n*n*(size_t)sizeof(int)); // current spin lattice
  //Leak check
  if(old==NULL || current == NULL){
      printf("Problem at memory allocation at host \n");
        exit(0);
      }

  int * d_old;
  int * d_current;
  double * d_w;// na valw void ** skatakia
  int *d_flag ;
  int flag ;
  //Host memory allocation and leak check
  if( hipMalloc((void **)&d_old ,n*n*(size_t)sizeof(int)) != hipSuccess  || hipMalloc((void **)&d_current,n*n*(size_t)sizeof(int))   != hipSuccess   || hipMalloc((void **)&d_w, WSIZE*WSIZE*(size_t)sizeof(double))   != hipSuccess || hipMalloc(&d_flag,(size_t)sizeof(int)) !=hipSuccess){
    printf("Problem at memory allocation");
    exit(0);
  }
  //Copy memory from host to device
  hipMemcpy(d_w, w, WSIZE*WSIZE*sizeof(double), hipMemcpyHostToDevice );
  hipMemcpy(d_old, G, n*n*sizeof(int), hipMemcpyHostToDevice );


  // run for k steps
  for(int l=0; l<k; l++){

    flag = 0;

    kernel2D<<<grid,block>>>(d_current, d_old, d_w, n  , d_flag );
    hipDeviceSynchronize();

  //cudaMemcpy(old, d_old, n*n*sizeof(int), cudaMemcpyDeviceToHost );
    hipMemcpy(current, d_current, n*n*sizeof(int), hipMemcpyDeviceToHost );
    // save result in G
    memcpy(G , current , n*n*sizeof(int));

    // swap the pointers for the next iteration
    swapElement(&d_old,&d_current);

    hipMemcpy(&flag , d_flag , (size_t)sizeof(int), hipMemcpyDeviceToHost);
    // terminate if no changes are made
    if(flag){
        printf("terminated: spin values stay same (step %d)\n" , l );
      exit(0);
    }
  }
  //Memory deallocation
  free(old);
  free(current);
  hipFree(d_old);
  hipFree(d_current);
  hipFree(d_w);
}

//Helper function to swap the pointers of the arrays
void swapElement(int  ** one, int  ** two) {
  int  * temp = * one;
  * one = * two;
  * two = temp;
}

 //The kernel function that updates the values of the ising model
__global__
void kernel2D(int *d_current, int *d_old, double *d_w, int n , int * d_flag)
{

  double influence = 0;
  // Compute global  column and row indices.
  int r = blockIdx.x * blockDim.x + threadIdx.x;
  int c = blockIdx.y * blockDim.y + threadIdx.y;

  // Check if within bounds.
  if ((c >= n) || (r >= n))
  return;

  for(int i = r; i<n; i+=blockDim.x*gridDim.x){
    for(int j = c; j<n; j+=blockDim.y*gridDim.y){

      for(int ii=0; ii<5; ii++){
        for(int jj=0; jj<5; jj++){
          influence +=  d_w(ii,jj) * d_old((i-2+n+ii)%n, (j-2+n+jj)%n, n);
        }
      }
      // magnetic moment gets the value of the SIGN of the weighted influence of its neighbors
      if(fabs(influence) < 10e-7){
        d_current(i,j,n) = d_old(i,j,n); // remains the same in the case that the weighted influence is zero
      }
      else if(influence > 10e-7){
        d_current(i,j,n) = 1;
        *d_flag = 0;
      }
      else if(influence < 0){
        d_current(i,j,n) = -1;
        *d_flag=0;
      }
      influence = 0;
    }
  }
}
